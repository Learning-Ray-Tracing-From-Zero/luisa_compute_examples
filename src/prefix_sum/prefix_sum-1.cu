#include <cstdio>
#include <hip/hip_runtime.h>


__global__ void init_array(int* arr) {
    if (blockDim.x * blockIdx.x + threadIdx.x == 0) {
        arr[0] = 3;
        arr[1] = 1;
        arr[2] = 7;
        arr[3] = 0;
        arr[4] = 4;
        arr[5] = 1;
        arr[6] = 6;
        arr[7] = 3;
    }
}

__global__ void kernel(int* arr, int n, int d) {
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    int step = 1 << (d - 1);
    if (k >= step && k < n) {
        arr[k] = arr[k - step] + arr[k];
    }
}


int main() {
    int n = 8;
    int *arr;
    hipMallocManaged(&arr, n * sizeof(int));

    init_array<<<1, 1>>>(arr);
    hipDeviceSynchronize();
    for (int i = 0; i < n; i++) { printf("%d ", arr[i]); }
    printf("\n");

    for (int d = 1; d <= static_cast<int>(log2(n)); ++d) {
        kernel<<<1, 8>>>(arr, n, d);
        hipDeviceSynchronize();
        for (int i = 0; i < n; i++) { printf("%d ", arr[i]); }
        printf("\n");
    }

    hipFree(arr);
    return 0;
}
