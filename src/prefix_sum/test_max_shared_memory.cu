#include <iostream>
#include <hip/hip_runtime.h>


int main() {
    int device_id = 0;
    hipDeviceProp_t props;
    if (hipError_t error = hipGetDeviceProperties(&props, device_id); error != hipSuccess) {
        std::cerr << "hipGetDeviceProperties failed: " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    std::cout << "Maximum shared memory per block on device " << device_id << ": "
              << props.sharedMemPerBlock / 1024.0 << " KB" << std::endl;

    return 0;
}
