#include <cstdio>
#include <hip/hip_runtime.h>


__global__ void init_array(int* arr) {
    if (blockDim.x * blockIdx.x + threadIdx.x == 0) {
        arr[0] = 3;
        arr[1] = 1;
        arr[2] = 7;
        arr[3] = 0;
        arr[4] = 4;
        arr[5] = 1;
        arr[6] = 6;
        arr[7] = 3;
    }
}

__global__ void block_prefix_sum(int* arr, int n) {
    int thread_x = threadIdx.x;
    for (int d = 1; d <= ceil(log2(n)); ++d) {
        int step = 1 << (d - 1);
        if (thread_x >= step && thread_x < n) {
            arr[thread_x] += arr[thread_x - step];
        }
        __syncthreads();
    }
}


int main() {
    int n = 8;
    int *arr;
    hipMallocManaged(&arr, n * sizeof(int));

    init_array<<<1, 1>>>(arr);
    hipDeviceSynchronize();
    for (int i = 0; i < n; i++) { printf("%d ", arr[i]); }
    printf("\n");

    block_prefix_sum<<<1, 8>>>(arr, n);
    hipDeviceSynchronize();
    for (int i = 0; i < n; i++) { printf("%d ", arr[i]); }

    hipFree(arr);
    return 0;
}
