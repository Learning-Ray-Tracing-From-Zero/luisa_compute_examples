#include <cstdio>
#include <hip/hip_runtime.h>


__global__ void init_array(int* arr) {
    if (blockDim.x * blockIdx.x + threadIdx.x == 0) {
        arr[0] = 3;
        arr[1] = 1;
        arr[2] = 7;
        arr[3] = 0;
        arr[4] = 4;
        arr[5] = 1;
        arr[6] = 6;
        arr[7] = 3;
    }
}

__global__ void block_prefix_sum(int* arr, int n) {
    int thread_x = threadIdx.x;
    for (int offset = 1; offset < n; offset *= 2) {
        int index = thread_x + offset;
        if (index < n) { arr[index] += arr[thread_x]; }
        __syncthreads();
    }
}


int main() {
    int n = 8;
    int *arr;
    hipMallocManaged(&arr, n * sizeof(int));

    init_array<<<1, 1, 1>>>(arr);
    hipDeviceSynchronize();
    for (int i = 0; i < n; i++) { printf("%d ", arr[i]); }
    printf("\n");

    block_prefix_sum<<<1, 8>>>(arr, n);
    hipDeviceSynchronize();
    for (int i = 0; i < n; i++) { printf("%d ", arr[i]); }

    hipFree(arr);
    return 0;
}
